#include "hip/hip_runtime.h"
#include "Renderer.cuh"


__global__ void renderoneray(Vector3f * framebuffer ,  int image_width, int image_height   , int turn_try)
{
	//blockIdx代表block的索引,blockDim代表block的大小，threadIdx代表thread线程的索引，因此对于一维的block和thread索引的计算方式如下
    __shared__ int m = 0; 
    int i =  blockIdx.x;    //blocksize , 最上层的执行
    int j =  threadIdx.x;   //blockdim ， 下面一层
    
    int i_circ = image_width/gridDim.x; // 有多少次循环
    int j_circ = image_height/blockDim.x; //x

    int i_left = image_width%gridDim.x; // 有多少次循环
    int j_left = image_height%blockDim.x; //x
    
    for(int i1  = i;  i1 < image_width; i1+=  gridDim.x)
    {
        for(int i2  = j;  i2 < image_height; i2+=  blockDim.x)
        {
            Ray camRay = scene.camera->generateRay(Vector2f(i1, i2));
            Vector3f RayC(1,1,1);
            Vector3f color(0,0,0);
             for(int turn = 0; turn < turn_try ;  ++turn)
            {
                color += scene.castRay(camRay, RayC , 0)/(float)turn_try;   //这条光线的追踪效果
            }
            framebuffer[i2*image_width + i1] +=  color;
            m++;
            UpdateProgress(m / (float)image_width* (float)image_height);
        }
    }
        
}



void renderall(std::vector<Vector3f> framebuffer ,  int image_width, int image_height   , int turn_try)
{
    assert(image_width * image_height == framebuffer.size());
    int device = 0;//设置使用第0块GPU进行运算
    hipSetDevice(device);//设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);//获取对应设备属性
    int threadMaxSize = devProp.maxThreadsPerBlock > image_height ? image_height : devProp.maxThreadsPerBlock  ;//每个线程块的最大线程数
    int blockSize =  devProp.maxGridSize > image_width ? image_width :  devProp.maxGridSize;//计算Block大小,block一维度是最大的，一般不会溢出
    dim3 thread(threadMaxSize);//设置thread
    dim3 block(blockSize);//设置block
    int size = length * sizeof(float);//计算空间大小
    float *sum =(float *)malloc(size) ;//开辟动态内存空间
    //开辟显存空间
    int *image_widthGPU,*image_heightGPU , *turn_tryGPU;
    hipMalloc((void**)&image_widthGPU,sizeof(int));
    hipMalloc((void**)&image_heightGPU,sizeof(int));
    hipMalloc((void**)&turn_tryGPU,sizeof(int));
    //内存->显存
    hipMemcpy((void*)image_widthGPU,(void*)image_width,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy((void*)image_heightGPU,(void*)image_height,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy((void*)turn_tryGPU,(void*)turn_try,sizeof(int),hipMemcpyHostToDevice);
    //运算

    Vector3f * framebuffer_GPU;
    cutilSafeCall( hipMalloc((void**) &framebuffer_GPU, sizeof(Vector3f) * framebuffer.size()));  // 申请显存空间
    cutilSafeCall( hipMemset(framebuffer_GPU, &framebuffer[0] , sizeof(Vector3f) * framebuffer.size() ,  hipMemcpyHostToDevice));
 

     
    renderoneray<<<block,thread>>>(framebuffer_GPU, *image_widthGPU , *image_heightGPU , *turn_tryGPU);
    //hipDeviceSynchronize();
    //显存->内存

    Vector3f * framebuffer_CPU;
    framebuffer_CPU = (Vector3f*)calloc(framebuffer.size(), sizeof(Vector3f)); 
    hipMemcpy(framebuffer_CPU, framebuffer_GPU, sizeof(Vector3f) * framebuffer.size() , hipMemcpyDeviceToHost);
    
    
    for(uint32_t i = 0; i <  framebuffer.size(); ++i)   // 打印Host端data
    {
        framebuffer[i] = framebuffer_CPU[i];
    }
    //释放显存
    hipFree(framebuffer_GPU);
    hipFree(image_widthGPU);
    hipFree(image_heightGPU);
    hipFree(turn_tryGPU);
    free(framebuffer_CPU);
}
 